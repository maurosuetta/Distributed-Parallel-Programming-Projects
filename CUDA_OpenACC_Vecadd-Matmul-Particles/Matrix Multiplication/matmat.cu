#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define BLOCKSIZE 16

// CUDA ERROR CHECK
#define CUDA_CHECK(call)                                          \
    do                                                            \
    {                                                             \
        hipError_t err = (call);                                 \
        if (err != hipSuccess)                                   \
        {                                                         \
            fprintf(stderr, "CUDA error at %s:%d: %s\n",          \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(1);                                              \
        }                                                         \
    } while (0)

void matmat_seq(double *A, double *B, double *C, const int N)
{
    double sum = 0.0;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            sum = 0.0;
            for (int k = 0; k < N; k++)
            {
                sum += A[i * N + k] * B[k * N + j];
            }
            C[i * N + j] = sum;
        }
    }
}


__global__ void matmat_naive_kernel(double *A, double *B, double *C, const int N)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    if (i < N && j < N)
    {
        double sum = 0.0;
        for (int k = 0; k < N; k++)
        {
            sum += A[i * N + k] * B[k * N + j];
        }
        C[i * N + j] = sum;
    }
}

__global__ void matmat_shared_kernel(double *A, double *B, double *C, const int N)
{
    int i = threadIdx.y + blockDim.y * blockIdx.y;
    int j = threadIdx.x + blockDim.x * blockIdx.x;

    __shared__ double A_s[BLOCKSIZE][BLOCKSIZE];
    __shared__ double B_s[BLOCKSIZE][BLOCKSIZE];

    double sum = 0.0;
    int blocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
    // Loop in tiles
    for (int block = 0; block < blocks; block++)
    {
        // Load A block to shared memory
        int k = block * BLOCKSIZE + threadIdx.x;
        if (i < N && k < N)
        {
            A_s[threadIdx.y][threadIdx.x] = A[i * N + k];
        }
        else
        {
            A_s[threadIdx.y][threadIdx.x] = 0.0;
        }

        // Load B block to shared memory
        k = block * BLOCKSIZE + threadIdx.y;
        if (k < N && j < N)
        {
            B_s[threadIdx.y][threadIdx.x] = B[k * N + j];
        }
        else
        {
            B_s[threadIdx.y][threadIdx.x] = 0.0;
        }
        // Wait for all threads to load A_s and B_s
        __syncthreads();

        // Compute the block
        // We don't need to check outbounds here
        // because if we are out, the Element of A_s or B_s is 0.0
        for (int kk = 0; kk < BLOCKSIZE; kk++)
        {
            sum += A_s[threadIdx.y][kk] * B_s[kk][threadIdx.x];
        }
        __syncthreads();
    }
    if (i < N && j < N)
        C[i * N + j] = sum;
}

void validation(double *h_C, double *C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            double err = fabs(h_C[i * N + j] - C[i * N + j]);
            if (err > 1.0e-6)
            {
                printf("Error at C[%d][%d]: fabs( %f - %f ) = %e > %e\n", i, j, h_C[i * N + j], C[i * N + j], err, 1.0e-6);
                exit(1);
            }
        }
    }
}

void copy_A_B_H2D(double *h_A, double *h_B, double *d_A, double *d_B, const size_t bytes,
                  hipEvent_t *event_start, hipEvent_t *event_end, float *total_time_ms, const char *case_name)
{
    float time_ms = 0.0;
    CUDA_CHECK(hipEventRecord(*event_start));

    CUDA_CHECK(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice));

    CUDA_CHECK(hipEventRecord(*event_end));
    CUDA_CHECK(hipEventSynchronize(*event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, *event_start, *event_end));
    printf("%s GPU H2D copy time: %.9f seconds\n", case_name, time_ms / 1000);
    *total_time_ms += time_ms;
}

void copy_C_D2H(double *h_C, double *d_C, const size_t bytes,
                hipEvent_t *event_start, hipEvent_t *event_end, float *total_time_ms, const char *case_name)
{
    float time_ms = 0.0;
    CUDA_CHECK(hipEventRecord(*event_start));

    CUDA_CHECK(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost));

    CUDA_CHECK(hipEventRecord(*event_end));
    CUDA_CHECK(hipEventSynchronize(*event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, *event_start, *event_end));
    printf("%s GPU D2H copy time: %.9f seconds\n", case_name, time_ms / 1000);
    *total_time_ms += time_ms;
}

void init_C_gpu(double *h_C, double *d_C, const int N)
{
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            h_C[i * N + j] = -1.0;
        }
    }

    CUDA_CHECK(hipMemset(d_C, 0, N * N * sizeof(double)));
}

int main(int argc, char *argv[])
{
    // Argument parsing
    if (argc != 3)
    {
        printf("Usage: %s <matrix size NxN> <check>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int check = atoi(argv[2]);

    printf("Matrix size: %d x %d\n", N, N);

    //
    // Memory allocation
    //
    // Host
    size_t bytes = N * N * sizeof(double);
    double *h_A = (double *)malloc(bytes);
    double *h_B = (double *)malloc(bytes);
    double *h_C = (double *)malloc(bytes);
    double *C = (double *)malloc(bytes);

    // Device
    double *d_A, *d_B, *d_C;
    CUDA_CHECK(hipMalloc((void **)&d_A, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_B, bytes));
    CUDA_CHECK(hipMalloc((void **)&d_C, bytes));
    CUDA_CHECK(hipMemset(d_C, 0, bytes)); // Init d_C to 0

    //
    // Matrices initialization
    //
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            // Row-major
            h_A[i * N + j] = drand48();
            h_B[i * N + j] = drand48();
            h_C[i * N + j] = -1.0;
            C[i * N + j] = -1.0;
        }
    }

    //
    // Sequential
    //
    if (check)
    {
        struct timespec start, end;
        clock_gettime(CLOCK_MONOTONIC, &start);

        matmat_seq(h_A, h_B, C, N);

        clock_gettime(CLOCK_MONOTONIC, &end);
        double elapsed = (end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec) / 1.0e9;
        printf("Sequential elapsed time: %.9f seconds\n", elapsed);
    }
    else
    {
        printf("Sequential and validation deactivated\n");
    }

    //
    // GPU computations
    //
    hipEvent_t event_start, event_end;
    float time_ms = 0.0;
    float total_time_ms = 0.0;
    CUDA_CHECK(hipEventCreate(&event_start));
    CUDA_CHECK(hipEventCreate(&event_end));

    //
    // Naive kernel
    //
    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "Naive");

    // Kernel launch
    dim3 threads(BLOCKSIZE, BLOCKSIZE);
    dim3 blocks((N + BLOCKSIZE - 1) / BLOCKSIZE, (N + BLOCKSIZE - 1) / BLOCKSIZE);
    CUDA_CHECK(hipEventRecord(event_start));

    matmat_naive_kernel<<<blocks, threads>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("Naive GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "Naive");

    printf("Naive GPU total time: %.9f seconds\n", total_time_ms / 1000);
    total_time_ms = 0.0;

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // Shared memory kernel
    //
    init_C_gpu(h_C, d_C, N);
    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "Shared");
    
    // Kernel launch
    CUDA_CHECK(hipEventRecord(event_start));
    matmat_shared_kernel<<<blocks, threads>>>(d_A, d_B, d_C, N);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("Shared GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "Shared");

    printf("Shared GPU total time: %.9f seconds\n", total_time_ms / 1000);
    total_time_ms = 0.0;

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // cuBLAS
    //
    init_C_gpu(h_C, d_C, N);
    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    // Copy Host to Device
    copy_A_B_H2D(h_A, h_B, d_A, d_B, bytes, &event_start, &event_end, &total_time_ms, "cuBLAS");

    CUDA_CHECK(hipEventRecord(event_start));

    const double alpha = 1.0;
    const double beta = 0.0;
    hipblasDgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A, N, &beta, d_C, N);

    CUDA_CHECK(hipDeviceSynchronize());

    CUDA_CHECK(hipEventRecord(event_end));
    CUDA_CHECK(hipEventSynchronize(event_end));
    CUDA_CHECK(hipEventElapsedTime(&time_ms, event_start, event_end));
    printf("cuBLAS GPU kernel time: %.9f seconds\n", time_ms / 1000);
    total_time_ms += time_ms;
    time_ms = 0.0;

    // Copy Device to Host
    copy_C_D2H(h_C, d_C, bytes, &event_start, &event_end, &total_time_ms, "cuBLAS");

    printf("cuBLAS GPU total time: %.9f seconds\n", total_time_ms / 1000);

    // Validate
    if (check)
        validation(h_C, C, N);

    //
    // Free memory
    //
    // Host
    free(h_A);
    free(h_B);
    free(h_C);
    free(C);

    // Device
    CUDA_CHECK(hipFree(d_A));
    CUDA_CHECK(hipFree(d_B));
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipEventDestroy(event_start));
    CUDA_CHECK(hipEventDestroy(event_end));
    hipblasDestroy(cublas_handle);

    return 0;
}